#include "hip/hip_runtime.h"
#include <sys/types.h>
#include <dirent.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <vector>
#include <algorithm>
#include <cstdio>
#include <ctime>
#include <fstream>
#include <string>
#include <cerrno>

#include <hipcub/hipcub.hpp>

#include "gputimer.h"
#include "stopwords.h"


using namespace std;



struct myVec
{
	char a[4096];
	int b;
};

struct posting
{
    unsigned long long key;
    int value;
};


__shared__ char sh_document[4097];
//__shared__ char sh_check[4097];
//__shared__ long sh_hash_val[4097];
__shared__ unsigned long long sh_tokens[4097];



__device__ int len(int i)
{
	int len = 0;
	for(;sh_tokens[i]==0;i++,len++);
	return len;
}


__device__ __host__ unsigned long SDBM(const char* str, unsigned int length)
{
	unsigned long hash = 0;
	unsigned int i;
	for(i = 0; i < length; ++str, i++)
	{
		hash = (*str) + (hash << 6) + (hash << 16) - hash;
	}
	return hash;
}

__device__ bool isAlpha(char ch)
{
	if(((ch>='a')&&(ch<='z'))||((ch>='A')&&(ch<='Z')))
		return true;
	else
    	return false;
}



__global__ void map(myVec *text,posting *d_out,unsigned long long int  *tell)
{ 
   
	//copying the document to shared memory
	int16_t t = threadIdx.x;
	int i=0;
	while((t<text[blockIdx.x].b)&&(t<4096))
	{
		sh_document[t] = text[blockIdx.x].a[t];
		//sh_check[t] = 0;
		sh_tokens[t] = 0;
		//sh_hash_val[t] = 0;
		i++;
		t = i*blockDim.x+threadIdx.x;
	}
	__syncthreads();
	sh_document[4096] = 0;
    //sh_check[4096] = 0;
	sh_tokens[4096] = 0;



    //Tokenising the document
    t = threadIdx.x;
    i = 0;
    while((t<text[blockIdx.x].b)&&(t<4097)&&(t!=0))
    {
    	if((isAlpha(sh_document[t]))&&(t!=0))
    	{
    		/*if((sh_document[t]>=65)&&(sh_document[t]<=90))
    			sh_document[i] = sh_document[i] + 32;*/
    		if((!isAlpha(sh_document[t-1]))&&(sh_tokens[t-1]==0))
    		{
    			atomicAdd(&sh_tokens[t-1],t-1);
    			
    		}
    		if(!isAlpha(sh_document[t+1])&&(t<4095)&&(sh_tokens[t+1]==0))
    		{
    			atomicAdd(&sh_tokens[t+1],t+1);
    			
    		}
    	}
    	else if(t!=0)
    	{
    		if(!isAlpha(sh_document[t-1])&&!isAlpha(sh_document[t+1])&&(sh_tokens[t]==0))
    		{
    			atomicAdd(&sh_tokens[t],t);
    		}
    	}
    	i++;
    	t = i*blockDim.x+threadIdx.x;
    }
    __syncthreads();

  
    // Creating the hash
    t = threadIdx.x;
    i = 0;
    while((t<text[blockIdx.x].b)&&(t<4097)&&(t!=0))
    {
    	
        if((sh_tokens[t]==0)&&(sh_tokens[t-1]!=0))
        	sh_tokens[t] = SDBM(&sh_document[t],len(t));
        else
        	sh_tokens[t] = 0;
    	i++;
    	t = i*blockDim.x+threadIdx.x;
    }
    __syncthreads();

    
    //Removing the stopwords
    t = threadIdx.x;
    i = 0;
    while((t<4097))
    {
        if(sh_tokens[i]!=0)
        	for(int j=0;j<174;j++)
        	{
        		/*if(sh_tokens[i]==d_stopwords[j])
        			atomicSub(&sh_tokens[i],sh_tokens[i]);//*/
        		atomicCAS(&sh_tokens[i],d_stopwords[j],0);
        	}
    	i++;
    	t = i*blockDim.x+threadIdx.x;
    }
    __syncthreads();



    /*//sorting algorithm
    if(threadIdx.x==0)
    {
    	for(int i = 0; i < 4097; i++)
    	{
    		for(int j = 0; j < 4097-i-1; j++)
    		{
    			if(sh_tokens[j]>sh_tokens[j+1])
    			{
    				unsigned long long temp = sh_tokens[j];
    				sh_tokens[j] = sh_tokens[j+1];
    				sh_tokens[j+1] = temp;
    			}
    		}
    	}
    }//*/

    //creating the postings
    if(blockIdx.x<588){
    t = threadIdx.x;
    i = 0;
    while(t<4097)
    {
        d_out[blockIdx.x*4097+t].key = sh_tokens[t];
        d_out[blockIdx.x*4097+t].value = blockIdx.x;
    	i++;
    	t = i*blockDim.x+threadIdx.x;
    }
    __syncthreads();
    }//*/


    /*int count = 0;
	unsigned long long t = ptr[threadIdx.x];
	__syncthreads();

	for(int i = 0; i < len; i++){
		if(ptr[i]<t)
			count++;
	}
	__syncthreads();
	ptr[count] = t;
    */



    
   /*   -----------------------needed later---------------------------  
   t = threadIdx.x;
    i=0;
    while((t<text[blockIdx.x].b)&&(t<4097))
	{
		if((sh_tokens[t]==0)&&(sh_document[t] != '\n')){
		sh_check[t] = sh_document[t];}
		else if(sh_tokens[t-1]==0)
			sh_check[t] = '\0';
		else
			sh_check[t] = '\0';
		i++;
		t = i*blockDim.x+threadIdx.x;
	}
	__syncthreads();
	------------------------------------------------------------------------*/

    
   ///*--------------------------------------------for checking values in shared memory------------------------------
	if((threadIdx.x==0)&&(blockIdx.x==0))
	{
		for(int i = 0; i < 4097;i++)
			tell[i] = sh_tokens[i];

	}
    //-------------------------------------------------------------------------------------------------------------*/
}

void read_directory(const string& name, vector<string>& v)
{
    DIR* dirp = opendir(name.c_str());
    struct dirent * dp;
    while ((dp = readdir(dirp)) != NULL) {
    	if(dp->d_name[0]=='.')
    		continue;
        v.push_back(dp->d_name);
    }
    closedir(dirp);
}


string get_file_content(const char* filename)
{
	ifstream in(filename, std::ios::in | std::ios::binary);
	if (in)
	{
		std::string contents;
		in.seekg(0, std::ios::end);
		contents.resize(in.tellg());
		in.seekg(0, std::ios::beg);
		in.read(&contents[0], contents.size());
		in.close();
		return(contents);
	}
	throw(errno);
}

int main(int argc,char** argv)
{
	unsigned long long int count;
	string path = "/home/aniket/out";
	char ch='y';
	unsigned long long int  *tell;
	int fileNumber; 
	posting *d_out;
	myVec *d_fileContent;
	vector<string> fileList;
	vector<string> fileContent;
	int *d_docId;
	hipError_t ce;
	//cout<<"Enter the path to dataset:\n";
	//getline(cin,path);


	read_directory(path,fileList);

	//-----------------------------------------------to be use to debug problems with file lsiting-----------------------------
	//	for(vector<string>::iterator it=fileList.begin();it!=fileList.end();++it)
	//	{
	//		cout<<*it<<endl;
	//	}
	//-------------------------------------------------------------------------------------------------------------------------

     for(unsigned int i=0;i<fileList.size();i++)
		fileContent.push_back(get_file_content((path+"/"+fileList[i]).c_str()));  //skipped first two files


    

	cout << "No of files copied "<<fileContent.size()<<endl;


    hipMallocManaged((void**)&d_fileContent,(fileContent.size())*sizeof(myVec));
    //hipMallocManaged((void**)&t,24576*sizeof(int));
    hipMallocManaged((void**)&tell,4097*sizeof(unsigned long long int));
    ce = hipMallocManaged((void**)&d_out, 4096*fileList.size()*sizeof(posting));

    cout<<hipGetErrorString(ce)<<endl;

    
    for(int i=0;i<fileContent.size();i++)
    {
    	//d_fileContent[i].a = (char*)malloc(fileContent[i].size()*sizeof(char));
    	hipMallocManaged((void**)&tell[i],4096*sizeof(char));
    	d_fileContent[i].b = fileContent[i].size();
    	strcpy(d_fileContent[i].a,fileContent[i].c_str());
    }

    //hipMemcpyToSymbol(HIP_SYMBOL(d_stopwords), stopword, 174*sizeof(unsigned long long));;

     GpuTimer kernelTimer,sortTimer;
     kernelTimer.Start();
     map<<<fileContent.size(),128>>>(d_fileContent,d_out,tell);
     kernelTimer.Stop();
     ce = hipDeviceSynchronize();
     cout<<hipGetErrorString(ce)<<endl;


     
     //--------------------------------------------------------For sorting-----------------------------------------------------
    unsigned long long *d_keys_in;      //array to hold the hashed terms
    int *d_values_in;                   //array to hold the docId
    unsigned long long *d_keys_out;     //array to hold the sorted hashed terms
    int *d_values_out;                  //array to hold the corrosponding docId
    int num_items = 4096*fileList.size();           //Number of element to be sorted

    void *d_temp_storage = NULL;        //auxallary space required for the sorting algorithm
    size_t temp_storage_bytes = 0;      //Size of the auxallary storage


    //Alocating spaces for the above variables
    hipMallocManaged((void**)&d_values_out, num_items*sizeof(int));
    hipMallocManaged((void**)&d_keys_out, num_items*sizeof(unsigned long long));
    hipMallocManaged((void**)&d_values_in, num_items*sizeof(int));
    hipMallocManaged((void**)&d_keys_in, num_items*sizeof(unsigned long long));

    //copying the keys and values form the output of mapper to the input of array for sorting
    for(unsigned long long i = 0; i < num_items; i++)
    {
        d_keys_in[i] = d_out[i].key;
        d_values_in[i] = d_out[i].value;
    }
     
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
    d_keys_in, d_keys_out, d_values_in, d_values_out, num_items);

    //Alocating auxallary space
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    sortTimer.Start();
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
    d_keys_in, d_keys_out, d_values_in, d_values_out, num_items);
    hipDeviceSynchronize();//*/
    sortTimer.Stop();

    //------------------------------------------------------------------------------------------------------------------------




     if(ce==hipSuccess)
     	cout<<"Index Created Sucessfully...!!"<<endl;
     else
     {	
     	cout<<"Index creation failed: "<<endl;
     	exit(1);
     }

    
    
    /*-------------------------------------------------------to be used to check file content-----------------------------------
		while (ch == 'y')
		{
			cout << "Enter the file number to be printed: " << endl;
			cin >> fileNumber;
			if (fileNumber < fileContent.size())
			cout << d_fileContent[fileNumber].a;
			cout << "Continue..?(y/n): ";
			cin >> ch;
		}
    //---------------------------------------------------------------------------------------------------------------------------*/
	  
    //---------------------------------------------------------Printing the execution time----------------------------------------

        cout<<"\nKernel Execution time: "<<kernelTimer.Elapsed()<<" ms"<<endl;
        cout<<"\nSorting time:          "<<sortTimer.Elapsed()<<" ms"<<endl;
		cout<<"\n";

    //----------------------------------------------------------------------------------------------------------------------------







    
    //--------------------------------------------------performing the search operation-------------------------------------------

        string term;
        GpuTimer searchTimer;
        cout<<"Enter the search term: ";
        cin>>term;
        //std::transform(term.begin(),term.end(),term.begin(),::tolower);
        unsigned long long term_hash = SDBM(term.c_str(),term.size());
        searchTimer.Start();
        for(unsigned long long i = num_items/2; i < num_items; i++)
        {
            if(d_keys_out[i]==term_hash)
                cout<<fileList[d_values_out[i]]<<endl;
        }
        searchTimer.Stop();


    //----------------------------------------------------------------------------------------------------------------------------


	/*for(int i=0;i<4096;i++)
	{
		cout<<d_out[4096*550+i].key<<" ";
	}//*/

    cout<<"\nSearch time: "<<searchTimer.Elapsed()<<" ms"<<endl;

	//cout<<fileContent[0];
	cout<<"\n";
	hipFree(d_out);
	hipFree(d_fileContent);
    hipFree(d_values_in);
    hipFree(d_keys_in);
    hipFree(d_values_out);
    hipFree(d_keys_out);
	return 0;
}
